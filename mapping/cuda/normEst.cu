#include "hip/hip_runtime.h"
#include "normEst.h"

#include <iostream>
#include <cstdio>

#include "gpuErrchk.h"
#include "hip/hip_vector_types.h"

using namespace std;
using namespace Eigen;

//gauss where the width is in line with sigma, but integrating over the curve would not yield 1 as
//it should be as simple as possible
inline __device__ 
float gaussUnscaled(float x, float sigma) {
	return exp((-x * x) / (2.0f * sigma * sigma));
}

//the tenth attempt of creating normals.... it is sort of stupid since the normals are in the camera frame.
//maybe we do a transformation from camera space to world space in a different kernel
__global__ 
void cudaCalcNormals_kernel(hipSurfaceObject_t d_std_min_std, 
                            hipSurfaceObject_t points,
                            hipSurfaceObject_t normals,
                            int width, int height, float threshold) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x >= width || y >= height) {
		return;
	}

	float3 ver = make_float3(0, 0, 0);
	float3 hor = make_float3(0, 0, 0);
	float weights[3][3] = {{ 1,  2,  1},
	                       { 0,  0,  0},
	                       {-1, -2, -1}};
	float4 center;
	surf2Dread(&center, points, x * sizeof(float4), y);
	float4 sensor;
	surf2Dread(&sensor, d_std_min_std, x * sizeof(float4), y);
	float sigma = sensor.z * 0.2f; //the quantisation noise on the sensor

	for(int i = 0; i < 3; i++) {
		for(int j = 0; j < 3; j++) {
			int _x = x + j - 1;
			int _y = y + i - 1;
			if(_x >= width || _y >= height || _x < 0 || _y < 0) {
				continue;
			}
			float4 point;
			surf2Dread(&point, points, _x * sizeof(float4), _y);

			float w = gaussUnscaled(center.z - point.z, sigma);
			float vw = w * weights[i][j];//vertical weights
			float hw = w * weights[j][i];//hoizontal weights
			//TODO: test if this is right
			float4 dist = point - center;
			ver += vw * make_float3(dist.x, dist.y, dist.z);
			hor += hw * make_float3(dist.x, dist.y, dist.z);
		}
	}

	//create the normal from the cross product
	// TODO: (why couldn't i just use eigen for this)
	float3 cross;
	cross.x = ver.y * hor.z - ver.z * hor.y;
	cross.y = ver.z * hor.x - ver.x * hor.z;
	cross.z = ver.x * hor.y - ver.y * hor.x;

	float length = sqrt(cross.x * cross.x + cross.y * cross.y + cross.z * cross.z);
	if(length == 0) {
		float4 normal = make_float4(0, 0, -1.0f, 1.0f);
		surf2Dwrite(normal, normals, x * sizeof(float4), y);
	} else {
		length = 1.0f / length;
		float4 normal = make_float4(cross.x * length, cross.y * length, cross.z * length, 1.0f);
		surf2Dwrite(normal, normals, x * sizeof(float4), y);
	}
}

void cudaCalcNormals(hipSurfaceObject_t d_std_min_std, 
                     hipSurfaceObject_t points,
                     hipSurfaceObject_t normals,
                     int width, int height, float threshold) {

	dim3 block(32, 32);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

	cudaCalcNormals_kernel<<<grid, block>>>(d_std_min_std, points, normals, width,
	                                        height, threshold);

	hipDeviceSynchronize();
}

__global__ 
void calcPoints_kernel(hipSurfaceObject_t d_std_min_std, 
                      hipSurfaceObject_t points,
                      int width, int height, Vector4f fxycxy) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x >= width || y >= height) {
		return;
	}
	float4 depth;
	surf2Dread(&depth, d_std_min_std, x * sizeof(float4), y);

	float z = depth.x;
	float fx = fxycxy[0];
	float fy = fxycxy[1];
	float cx = fxycxy[2];
	float cy = fxycxy[3];
	float4 point;
	point.x = (float(x) - cx) * z * fx;
	point.y = (float(y) - cy) * z * fy;
	point.z = z;
	point.w = 1.0f;
	surf2Dwrite(point, points, x * sizeof(float4), y);
}


void cudaCalcPoints(hipSurfaceObject_t d_std_min_std, 
                    hipSurfaceObject_t points,
                    int width, int height, Vector4f fxycxy) {

	dim3 block(32,32);
	dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
	// Invert the last two elements so we don't have to do it in the kernel
	fxycxy[0] = 1.0f / fxycxy[0];
	fxycxy[1] = 1.0f / fxycxy[1];

	calcPoints_kernel<<<grid, block>>>(d_std_min_std, points, width, height, 
	                                   fxycxy);

	hipDeviceSynchronize();
}
