#include "hip/hip_runtime.h"
#include "test.h"

struct RGBStorage {
	unsigned char r;
	unsigned char g;
	unsigned char b;
};

__global__ 
void copy_kernel(const hipTextureObject_t texture,  
                 cv::cuda::PtrStepSz<RGBStorage> to) {

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x >= to.cols || y >= to.rows) {
		return;//out of bounds
	}

	RGBStorage point;

	float4 buff = tex2D<float4>(texture, x, y);

	point.r = buff.x * 255.0;
	if(x == 0) {
		//printf("%d\n",y);
	}
	point.g = buff.y * 255.0;
	point.b = buff.z * 255.0;

	to(y, x) = point;
}

void copy(hipTextureObject_t texture, cv::cuda::GpuMat &to) {
	dim3 block(32, 32);
	dim3 grid((to.cols + block.x - 1) / block.x, 
	          (to.rows + block.y - 1) / block.y);
	copy_kernel<<<grid, block>>>(texture, to);
	hipDeviceSynchronize();
}
